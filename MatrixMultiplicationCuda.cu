#include "hip/hip_runtime.h"
#include "common.h"
#include <hip/hip_runtime.h>
#include <chrono>

using namespace std;

void initializeMatrix(int *matrix, const int rows, const int cols){
  for (size_t i = 0; i < rows * cols; i++)
    matrix[i] = i;
}

__global__ void multiplyMatricesWithCuda(int *matrixA, int *matrixB, int *result, const int rows, const int cols){
  unsigned int ix = threadIdx.x + blockIdx.x * blockDim.x;
  unsigned int iy = blockIdx.y;
  if(ix < rows && iy < cols){
    int sum = 0;
    for (size_t i = 0; i < cols; i++) {
      sum += matrixA[iy * rows + i] * matrixB[i * cols + ix];
    }
    result[iy * rows + ix] = sum;
  }
}

int main(int argc, char const *argv[]) {
  // set up device
  int dev = 0;
  hipDeviceProp_t deviceProp;
  SAFE_CALL(hipGetDeviceProperties(&deviceProp, dev), "Error device prop");
  printf("Using Device %d: %s\n", dev, deviceProp.name);
  SAFE_CALL(hipSetDevice(dev), "Error setting device");

  // Declare matrices
  int *matrixA;
  int *matrixB;
  int *result;
  int *dev_matrixA;
  int *dev_matrixB;
  int *dev_result;

  // Set up size of matrix
  const int rows = 1000;
  const int cols = 1000;
  printf("Matrix size: rows %d columns %d\n", rows, cols);

  int bytes = rows * cols * sizeof(int);

  // Allocate matrices memory
  matrixA = (int *) malloc(bytes);
  matrixB = (int *) malloc(bytes);
  result = (int *) malloc(bytes);

  // Initialize matrices
  initializeMatrix(matrixA, rows, cols);
  initializeMatrix(matrixB, rows, cols);

  // Allocate device global memory
  SAFE_CALL(hipMalloc((void **)&dev_matrixA, bytes), "Error allocating dev_matrixA");
  SAFE_CALL(hipMalloc((void **)&dev_matrixB, bytes), "Error allocating dev_matrixB");
  SAFE_CALL(hipMalloc((void **)&dev_result, bytes), "Error allocating dev_result");

  // Transfer data from host to device
  SAFE_CALL(hipMemcpy(dev_matrixA, matrixA, bytes, hipMemcpyHostToDevice), "Error copying dev_matrixA");
  SAFE_CALL(hipMemcpy(dev_matrixB, matrixB, bytes, hipMemcpyHostToDevice), "Error copying dev_matrixB");

  // Invoke kernel at host side
  int dimx = 512;
  dim3 block(dimx, 1);
  dim3 grid((rows + block.x - 1) / block.x, cols);

  auto start_at = chrono::high_resolution_clock::now();
  multiplyMatricesWithCuda<<<grid, block>>>(dev_matrixA, dev_matrixB, dev_result, rows, cols);
  SAFE_CALL(hipDeviceSynchronize(), "Error executing kernel");
  auto end_at = chrono::high_resolution_clock::now();
  chrono::duration<float, std::milli> duration_ms = end_at - start_at;

  printf("Multiply matrices on GPU <<<(%d,%d), (%d,%d)>>> elapsed: %f ms (%.2f seconds)\n",
        grid.x, grid.y, block.x, block.y, duration_ms.count(), duration_ms.count() / 1000);

  // SAFE_CALL kernel error
  SAFE_CALL(hipGetLastError(), "Error with last error");

  // Copy kernel result back to host side
  SAFE_CALL(hipMemcpy(result, dev_result, bytes, hipMemcpyDeviceToHost), "Error copying dev_result");

  // Free device global memory
  SAFE_CALL(hipFree(dev_matrixA), "Error freeing memory");
  SAFE_CALL(hipFree(dev_matrixB), "Error freeing memory");
  SAFE_CALL(hipFree(dev_result), "Error freeing memory");

  // Free matrices memory
  free(matrixA);
  free(matrixB);
  free(result);

  // Reset device
  SAFE_CALL(hipDeviceReset(), "Error reseting");

  return 0;
}
